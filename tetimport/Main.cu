#include "hip/hip_runtime.h"
#define NO_MSG
#include "tetgen_io.h"

const int width = 320, height=240;




RGB trace(Ray r, tetrahedra_mesh *mesh, int32_t start, int depth)
{

	rayhit firsthit;
	traverse_ray(mesh, r, start, firsthit);

	face fc = mesh->get_face(firsthit.face);
	node a1 = mesh->get_node(fc.node_a);
	node a2 = mesh->get_node(fc.node_b);
	node a3 = mesh->get_node(fc.node_c);
	double d = intersect_dist(r, a1.f_node(), a2.f_node(), a3.f_node());


	return RGB(0, 0, (int)d*5); // abh�ngig von d
}




int main()
{
	// load tetrahedral model
	tetrahedra_mesh tetmesh;
	tetmesh.load_tet_ele("untitled.1.ele");
	tetmesh.load_tet_neigh("untitled.1.neigh");
	tetmesh.load_tet_node("untitled.1.node");
	tetmesh.load_tet_face("untitled.1.face");
	tetmesh.load_tet_t2f("untitled.1.t2f");
	tetmesh.cam.d = make_float4(0, 1, 0, 0);
	tetmesh.cam.o = make_float4(0, 5, 5, 0);
	tetmesh.curr = tetmesh.cam;
	

	// Get bounding box
	tetmesh.init_BBox();
	fprintf_s(stderr, "\nBounding box:MIN xyz - %f %f %f \n", tetmesh.boundingbox.min.x, tetmesh.boundingbox.min.y, tetmesh.boundingbox.min.z);
	fprintf_s(stderr, "             MAX xyz - %f %f %f \n\n", tetmesh.boundingbox.max.x, tetmesh.boundingbox.max.y, tetmesh.boundingbox.max.z);

	// Find camera tetrahedra
	int32_t start;
	start = tetmesh.GetTetrahedraFromPoint(tetmesh.cam.o);
	if (start == -1) {
		fprintf_s(stderr, "Camera point not inside mesh! Aborting\n");
		system("PAUSE");
		return 0;
	}	else fprintf_s(stderr, "Starting point (camera) tetra number: %lu\n\n",start);


	// raytracing stuff
	RGB *color=new RGB[width*height];
		for (int x = 0; x < width; x++){
			for (int y = 0; y < height; y++){

			tetmesh.curr.o = make_float4(0, 5, 5, 0);
			float4 cam = camcr(width, height, x, y);
			tetmesh.curr.d = normalize(cam - tetmesh.curr.o);

			color[(height - y - 1)*width + x] = trace(tetmesh.curr, &tetmesh, start, 0);
			//color[(height - y - 1)*width + x] = RGB(40, 0, 0); // auch hier jeder zweite y-wert weg
		}
	}

	// write to image
	FILE *f = fopen("image.ppm", "w");         // Write image to PPM file. 
	fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);
	for (int i = 0; i<width*height; i++)
		fprintf(f, "%d %d %d ", (int)color[i].x, (int)color[i].y, (int)color[i].z);
}


