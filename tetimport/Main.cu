#include "hip/hip_runtime.h"
#include "tetgen_io.h"
#include "cu_gl.h"
#include "cuPrintf.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int width = 800, height=600, spp = 8;



unsigned int WangHash(unsigned int a) {
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}



// CUDA error checking
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		system("PAUSE");
		if (abort) exit(code);
	}
}


__device__ float getDepth(Ray r, mesh2 *mesh, rayhit firsthit)
{
	float4 a1 = make_float4(mesh->n_x[mesh->f_node_a[firsthit.face]], mesh->n_y[mesh->f_node_a[firsthit.face]], mesh->n_z[mesh->f_node_a[firsthit.face]], 0);
	float4 a2 = make_float4(mesh->n_x[mesh->f_node_b[firsthit.face]], mesh->n_y[mesh->f_node_b[firsthit.face]], mesh->n_z[mesh->f_node_b[firsthit.face]], 0);
	float4 a3 = make_float4(mesh->n_x[mesh->f_node_c[firsthit.face]], mesh->n_y[mesh->f_node_c[firsthit.face]], mesh->n_z[mesh->f_node_c[firsthit.face]], 0);
	float c = abs(intersect_dist(r, a1, a2, a3));
	float k = ((255 - 0) / (0 - 80)); // in zweiter klammer erster wert ist untere grenze distanzwerte
	float d = 0 - (80 * k);
	return (c*k) + d;
}


__device__ RGB radiance(Ray r, mesh2 *mesh, int32_t start, int depth)
{
	rayhit firsthit;
	traverse_ray(mesh, r, start, firsthit, depth);
	float d2 = getDepth(r, mesh, firsthit); // return depth value
	RGB rd;
	rd.x = 0; rd.y = 0; rd.z = d2;
	return rd; 
}


__global__ void renderKernel(mesh2 *tetmesh, int32_t start, float4 cam_o, float4 cam_d, float4 cam_u, float4 *c, unsigned int hashedframenumber)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = (height - y - 1)*width + x;

	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedframenumber + threadId, 0, 0, &randState);

	// raytracing stuff
	RGB c0(0);

	for (int s = 0; s < spp; s++)
			{
		float yu = 1.0f - ((y + hiprand_uniform(&randState)) / float(height - 1));
		float xu = (x + hiprand_uniform(&randState)) / float(width - 1);
				Ray ray = makeCameraRay(45.0f, cam_o, cam_d, cam_u, xu, yu);
				RGB rd = radiance(ray, tetmesh, start, 0);
				c0 = c0 + rd;
			}
	c0 = c0 / 4;
	c[i] = make_float4( c0.x, c0.y, c0.z,0);
}


int main(int argc, char *argv[])
{
	int frames = 0;
	hipDeviceProp_t  prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	hipChooseDevice(&dev, &prop);

	tetrahedra_mesh tetmesh;
	tetmesh.load_tet_ele("test1.1.ele");
	tetmesh.load_tet_neigh("test1.1.neigh");
	tetmesh.load_tet_node("test1.1.node");
	tetmesh.load_tet_face("test1.1.face");
	tetmesh.load_tet_t2f("test1.1.t2f");



	
	// ===========================
	//     mesh2 testing
	// ===========================

	mesh2 *mesh;
	gpuErrchk(hipMallocManaged(&mesh, sizeof(mesh2)));

	// ELEMENT INDICES
	mesh->edgenum = tetmesh.edgenum;
	mesh->facenum = tetmesh.facenum;
	mesh->nodenum = tetmesh.nodenum;
	mesh-> tetnum = tetmesh.tetnum;

	// NODES - funktioniert
	hipMallocManaged(&mesh->n_index, mesh->nodenum*sizeof(uint32_t));
	for (auto i : tetmesh.nodes) mesh->n_index[i.index] = i.index;
	hipMallocManaged(&mesh->n_x, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_y, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_z, mesh->nodenum*sizeof(float));
	for (auto i : tetmesh.nodes) mesh->n_x[i.index] = i.x;
	for (auto i : tetmesh.nodes) mesh->n_y[i.index] = i.y;
	for (auto i : tetmesh.nodes) mesh->n_z[i.index] = i.z;

	// FACES
	hipMallocManaged(&mesh->f_index, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_index[i.index] = i.index;
	hipMallocManaged(&mesh->f_node_a, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_b, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_c, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_node_a[i.index] = i.node_a;
	for (auto i : tetmesh.faces) mesh->f_node_b[i.index] = i.node_b;
	for (auto i : tetmesh.faces) mesh->f_node_c[i.index] = i.node_c;
	hipMallocManaged(&mesh->face_is_constrained, mesh->facenum*sizeof(bool));
	hipMallocManaged(&mesh->face_is_wall, mesh->facenum*sizeof(bool));
	for (auto i : tetmesh.faces) mesh->face_is_constrained[i.index] = i.face_is_constrained;
	for (auto i : tetmesh.faces) mesh->face_is_wall[i.index] = i.face_is_wall;

	// TETRAHEDRA
	hipMallocManaged(&mesh->t_index, mesh->tetnum*sizeof(uint32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_index[i.number] = i.number;
	hipMallocManaged(&mesh->t_findex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_findex1[i.number] = i.findex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex2[i.number] = i.findex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex3[i.number] = i.findex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex4[i.number] = i.findex4;
	hipMallocManaged(&mesh->t_nindex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex1[i.number] = i.nindex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex2[i.number] = i.nindex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex3[i.number] = i.nindex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex4[i.number] = i.nindex4;
	hipMallocManaged(&mesh->t_adjtet1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet1[i.number] = i.adjtet1;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet2[i.number] = i.adjtet2;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet3[i.number] = i.adjtet3;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet4[i.number] = i.adjtet4;

	// ===========================
	//     mesh end
	// ===========================

	// Get bounding box
	BBox box = init_BBox(mesh);
	fprintf_s(stderr, "\nBounding box:MIN xyz - %f %f %f \n", box.min.x, box.min.y, box.min.z);
	fprintf_s(stderr, "             MAX xyz - %f %f %f \n\n", box.max.x, box.max.y, box.max.z);

	float4 cam_o = make_float4(-16, 5, -5, 0);
	float4 cam_d = make_float4(0, 0, 0, 0);
	float4 cam_u = make_float4(0, 0, 1, 0);

	float4 *cr;
	gpuErrchk(hipMallocManaged(&cr, width * height * sizeof(float4)));


	clock_t t1 = clock();
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);

	GetTetrahedraFromPoint << <mesh->tetnum, 1>> >(mesh, cam_o);
	gpuErrchk(hipDeviceSynchronize()); // kamera erfolgreich abgerufen..

	if (start == 0) 
	{
		fprintf(stderr, "Starting point outside tetrahedra! Aborting ... \n");
		system("PAUSE");
		abort;

	} else fprintf(stderr, "Starting tetrahedra - camera: %lu \n", start);
	

	renderKernel<<<grid,block>>>(mesh, start, cam_o, cam_d, cam_u, cr, WangHash(frames));
	//renderKernel << <1,1 >> >(mesh, start, cam_o, cam_d, cam_u, cr);
	gpuErrchk(hipDeviceSynchronize());

	clock_t t2 = clock();
	double t = (double)(t2 - t1) / CLOCKS_PER_SEC;
	printf("\nRender time: %fs.\n", t);

	FILE *f = fopen("image.ppm", "w");
	fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);
	for (int i = 0; i<width*height; i++)
		fprintf(f, "%d %d %d ", (int)cr[i].x, (int)cr[i].y, (int)cr[i].z);

	hipFree(mesh);
	hipFree(cr);


	system("PAUSE");;
}


