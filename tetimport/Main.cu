#include "hip/hip_runtime.h"
/*
*  tetrahedra-based raytracer
*  Copyright (C) 2015  Christian Lehmann
*
*  This program is free software; you can redistribute it and/or modify
*  it under the terms of the GNU General Public License as published by
*  the Free Software Foundation; either version 2 of the License, or
*  (at your option) any later version.
*
*  This program is distributed in the hope that it will be useful,
*  but WITHOUT ANY WARRANTY; without even the implied warranty of
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*  GNU General Public License for more details.
*/

#define GLEW_STATIC
#include "tetgen_io.h"
#include "cuPrintf.cuh"
#include ""
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v7.5\extras\CUPTI\include\GL\glew.h"
#include "GLFW/glfw3.h"
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int width = 640, height=480, spp = 1;
float3* cr;
float3* accumulatebuffer;
int frames = 0;
__device__ float gamma = 1.0f;
__device__ float fov = 40.0f;
BBox box;
GLuint vbo;
mesh2 *mesh;
#define MAX_DEPTH 3

// Camera
bool keys[1024];
GLfloat sensitivity = 0.15f;
bool firstMouse = true;
float4 cam_o = make_float4(-14, 11, 11, 0);
float4 cam_d = make_float4(0.1f, 0.1f, 0.1f, 0);
float4 cam_u = make_float4(0, 0, 1, 0);
GLfloat Yaw = 90.0f;	// horizontal inclination
GLfloat Pitch = 0.0f; // vertikal inclination
GLfloat lastX = width / 2.0; //screen center
GLfloat lastY = height / 2.0;
GLfloat deltaTime = 0.0f;	// Time between current frame and last frame
GLfloat lastFrame = 0.0f;

union Color  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};

// CUDA error checking
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		system("PAUSE");
		if (abort) exit(code);
	}
}

unsigned int WangHash(unsigned int a) {
	// richiesams.blogspot.co.nz/2015/03/creating-randomness-and-acummulating.html
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

static void error_callback(int error, const char* description)
{
	fputs(description, stderr);
}


void updateCamPos()
{
	CheckOutOfBBox(&box, cam_o);
	//look for new tetrahedra...
	uint32_t _dim = 2 + pow(mesh->tetnum, 0.25);
	dim3 Block(_dim, _dim, 1);
	dim3 Grid(_dim, _dim, 1);
	GetTetrahedraFromPoint << <Grid, Block >> >(mesh, cam_o);
	gpuErrchk(hipDeviceSynchronize());
}

static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
	// first value sets velocity
	GLfloat cameraSpeed = 0.5f * deltaTime;
	if (key >= 0 && key < 1024)
	{
		if (action == GLFW_PRESS)
		{
			keys[key] = true;
		}
		else if (action == GLFW_RELEASE)
		{
			keys[key] = false;
		}

	}
	if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
	{
		glfwSetWindowShouldClose(window, GL_TRUE);
	}
		
	if (keys[GLFW_KEY_A])
	{
		updateCamPos();
		cam_o -= normalizeCPU(CrossCPU(minus(cam_d, cam_o), cam_u)) * cameraSpeed;
	}
	if (keys[GLFW_KEY_D])
	{
		updateCamPos();
		cam_o += normalizeCPU(CrossCPU(minus(cam_d, cam_o), cam_u)) * cameraSpeed;
	}
	if (keys[GLFW_KEY_W])
	{
		updateCamPos();
		cam_o += minus(cam_d, cam_o) * cameraSpeed;
	}
	if (keys[GLFW_KEY_S])
	{
		updateCamPos();
		cam_o -= minus(cam_d, cam_o) * cameraSpeed;
	}
}

void mouse_callback(GLFWwindow* window, double xpos, double ypos)
{	
	if (firstMouse)
	{
		lastX = xpos;
		lastY = ypos;
		firstMouse = false;
	}
	GLfloat xoffset = xpos - lastX;
	GLfloat yoffset = lastY - ypos; // Reversed since y-coordinates go from bottom to left
	
	lastX = xpos;
	lastY = ypos;
	xoffset *= sensitivity;
	yoffset *= sensitivity;

	Yaw += yoffset; //ge�ndert - vorher y/x vertauscht
	Pitch += xoffset;

	// Make sure that when pitch is out of bounds, screen doesn't get flipped
	if (Pitch > 89.0f)
		Pitch = 89.0f;
	if (Pitch < -89.0f)
		Pitch = -89.0f;

	float4 front;
	front.x = cos(radian(Yaw)) * cos(radian(Pitch));
	front.y = sin(radian(Pitch));
	front.z = sin(radian(Yaw)) * cos(radian(Pitch));
	cam_d = normalizeCPU(front);
}



__device__ float getDepth(Ray r, mesh2 *mesh, int32_t face)
{
	float4 a1 = make_float4(mesh->n_x[mesh->f_node_a[face]], mesh->n_y[mesh->f_node_a[face]], mesh->n_z[mesh->f_node_a[face]], 0);
	float4 a2 = make_float4(mesh->n_x[mesh->f_node_b[face]], mesh->n_y[mesh->f_node_b[face]], mesh->n_z[mesh->f_node_b[face]], 0);
	float4 a3 = make_float4(mesh->n_x[mesh->f_node_c[face]], mesh->n_y[mesh->f_node_c[face]], mesh->n_z[mesh->f_node_c[face]], 0);
	float c = abs(intersect_dist(r, a1, a2, a3));
	float new_value = ((c - 0.0f) / (100.0f - 0.0f)) * (1.0f - 0.0f) + 0.0f; // assume max depth of 100, color conversion to 0-1 range
	return new_value;
}

__device__ float4 getTriangleNormal(const float4 &p1, const float4 &p2, const float4 &p3)
{
	return(Cross(p2 - p1, p3 - p1));
}

__device__ RGB visualizeDepth(Ray r, mesh2 *mesh, int32_t start, int depth)
{
	rayhit firsthit;
	traverse_ray(mesh, r, start, firsthit, depth);
	float d2 = getDepth(r, mesh, firsthit.face); // gets depth value

	RGB rd;
	if (firsthit.wall == true) { rd.x = 0.5; rd.y = 0.8; rd.z = 0.1; }
	if (firsthit.constrained == true) { rd.x = 0.1; rd.y = 0.1; rd.z = d2; }
	return rd; 
}


__device__ RGB radiance(mesh2 *mesh, int32_t &start, Ray &ray, hiprandState* randState)
{
	Ray r;
	r.d = ray.d;
	r.o = ray.o;

	float4 mask = make_float4(1.0f, 1.0f, 1.0f, 0.0f);	// colour mask
	float4 accucolor = make_float4(0.0f, 0.0f, 0.0f, 0.0f);	// accumulated colour
	int pd=0;

	for (int depth = 1; depth <= MAX_DEPTH; depth++)
	{
		float4 f;  // primitive colour
		float4 emit; // primitive emission colour
		float4 x; // intersection point
		float4 n; // normal
		float4 nl; // oriented normal
		float4 d; // ray direction of next path segment


		rayhit firsthit;
		traverse_ray(mesh, r, start, firsthit, pd);

		// set new starting tetrahedra and ray origin
		float4 a1 = make_float4(mesh->n_x[mesh->f_node_a[firsthit.face]], mesh->n_y[mesh->f_node_a[firsthit.face]], mesh->n_z[mesh->f_node_a[firsthit.face]], 0);
		float4 a2 = make_float4(mesh->n_x[mesh->f_node_b[firsthit.face]], mesh->n_y[mesh->f_node_b[firsthit.face]], mesh->n_z[mesh->f_node_b[firsthit.face]], 0);
		float4 a3 = make_float4(mesh->n_x[mesh->f_node_c[firsthit.face]], mesh->n_y[mesh->f_node_c[firsthit.face]], mesh->n_z[mesh->f_node_c[firsthit.face]], 0);
		// get intersection distance
		float t = intersect_dist(r, a1, a2, a3);

		x = r.o + r.d*t;  // intersection point
		n = normalize(getTriangleNormal(a1, a2, a3));  // normal 
		nl = Dot(n, r.d) < 0 ? n : n * -1;  // correctly oriented normal
		f = make_float4(0.3f, 0.4f, 0.1f, 0.0f);  // triangle colour
		emit = make_float4(0.1f, 0.1f, 0.1f, 0.0f);
		accucolor += (mask * emit);

		firsthit.refl = REFR;

		// ideal refraction (based on smallpt code by Kevin Beason)
		if (firsthit.refl == REFR){

			bool into = Dot(n, nl) > 0; // is ray entering or leaving refractive material?
			float nc = 1.0f;  // Index of Refraction air
			float nt = 1.5f;  // Index of Refraction glass/water
			float nnt = into ? nc / nt : nt / nc;  // IOR ratio of refractive materials
			float ddn = Dot(r.d, nl);
			float cos2t = 1.0f - nnt*nnt * (1.f - ddn*ddn);

			if (cos2t < 0.0f) // total internal reflection 
			{
				d = reflect(r.d, n); //d = r.dir - 2.0f * n * dot(n, r.dir);
				x += nl * 0.01f;
			}
			else // cos2t > 0
			{
				// compute direction of transmission ray
				float4 tdir = normalize(r.d * nnt - n * ((into ? 1 : -1) * (ddn*nnt + sqrtf(cos2t))));

				float R0 = (nt - nc)*(nt - nc) / (nt + nc)*(nt + nc);
				float c = 1.f - (into ? -ddn : Dot(tdir, n));
				float Re = R0 + (1.f - R0) * c * c * c * c * c;
				float Tr = 1 - Re; // Transmission
				float P = .25f + .5f * Re;
				float RP = Re / P;
				float TP = Tr / (1.f - P);

				// randomly choose reflection or transmission ray
				if (hiprand_uniform(randState) < 0.25) // reflection ray
				{
					mask *= RP;
					d = reflect(r.d, n);
					x += nl * 0.02f;
				}
				else // transmission ray
				{
					mask *= TP;
					d = tdir; //r = Ray(x, tdir); 
					x += nl * 0.0005f; // epsilon must be small to avoid artefacts
				}
			}
		}


		// ideal diffuse reflection (see "Realistic Ray Tracing", P. Shirley)
		if (firsthit.refl == DIFF){

			// create 2 random numbers
			float r1 = 2 * PI * hiprand_uniform(randState);
			float r2 = hiprand_uniform(randState);
			float r2s = sqrtf(r2);

			// compute orthonormal coordinate frame uvw with hitpoint as origin 
			float4 w = nl;
			float4 u = normalize(Cross((fabs(w.x) > .1 ? make_float4(0, 1, 0, 0) : make_float4(1, 0, 0, 0)), w));
			float4 v = Cross(w, u);

			// compute cosine weighted random ray direction on hemisphere 
			d = normalize(u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrtf(1 - r2));

			// offset origin next path segment to prevent self intersection
			x += nl * 0.03;

			// multiply mask with colour of object
			mask *= f;
		}


		if (firsthit.refl == SPEC)
		{
			// compute reflected ray direction according to Snell's law
			d = r.d - 2.0f * n * Dot(n, r.d);
			// offset origin next path segment to prevent self intersection
			x += nl * 0.01f;
			// multiply mask with colour of object
			mask *= f;
		}
		r.o = x;
		r.d = d; // new ray direction
		start = firsthit.tet; // new tet origin
	}
	RGB rgb;
	rgb.x = accucolor.x;
	rgb.y = accucolor.y;
	rgb.z = accucolor.z;
	return rgb;
}


__global__ void renderKernel(mesh2 *tetmesh, int32_t start, float4 cam_o, float4 cam_d, float4 cam_u, float3 *accumbuffer, float3 *c, unsigned int hashedframenumber, int framenumber)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = (height - y - 1)*width + x;

	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprandState randState;
	hiprand_init(hashedframenumber + threadId, 0, 0, &randState);


	RGB pixelcol(0);
	for (int s = 0; s < spp; s++)
	{
		float yu = 1.0f - ((y + hiprand_uniform(&randState)) / float(height - 1));
		float xu = (x + hiprand_uniform(&randState)) / float(width - 1);
		Ray ray = makeCameraRay(fov, cam_o, cam_d, cam_u, xu, yu);
		//RGB rd = visualizeDepth(ray, tetmesh, start, 0);
		pixelcol += radiance(tetmesh, start, ray, &randState)*(1. / spp);
	}

	accumbuffer[i] += pixelcol;
	float3 tempcol = accumbuffer[i] / framenumber;

	Color fcolour;
	float3 colour = make_float3(clamp(tempcol.x, 0.0f, 1.0f), clamp(tempcol.y, 0.0f, 1.0f), clamp(tempcol.z, 0.0f, 1.0f));

	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / gamma) * 255), (unsigned char)(powf(colour.y, 1 / gamma) * 255), (unsigned char)(powf(colour.z, 1 / gamma) * 255), 1);
	c[i] = make_float3(x, y, fcolour.c);
}


void render()
{
	GLFWwindow* window;
	if (!glfwInit()) exit(EXIT_FAILURE);
	window = glfwCreateWindow(width, height, "", NULL, NULL);
	glfwMakeContextCurrent(window);
	glfwSetErrorCallback(error_callback);
	glfwSetKeyCallback(window, key_callback);
	glfwSetCursorPosCallback(window, mouse_callback);
	glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

	glewExperimental = GL_TRUE;
	glewInit();
	if (!glewIsSupported("GL_VERSION_2_0 ")) 
	{
		fprintf(stderr, "GLEW not supported.");
		fflush(stderr);
		exit(0);
	}
	fprintf(stderr, "GLEW successfully initialized  \n");


	glClearColor(0.0, 0.0, 0.0, 0.0);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, width, 0.0, height, 0, 1);
	
	glGenBuffers(1, &vbo);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glBufferData(GL_ARRAY_BUFFER, width * height * sizeof(float3), 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	cudaGLRegisterBufferObject(vbo);
	fprintf(stderr, "VBO created  \n");
	fprintf(stderr, "Entering glutMainLoop...  \n");

	while (!glfwWindowShouldClose(window))
	{
		frames++;
		// Calculate deltatime of current frame
		GLfloat currentFrame = glfwGetTime();
		deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;
		std::stringstream title;
		title << "tetra_mesh (2015)   -   deltaTime: " << deltaTime*1000 << " ms. (16-36 optimal)";
		glfwSetWindowTitle(window, title.str().c_str());

		glClear(GL_COLOR_BUFFER_BIT);
		glfwPollEvents();
		cudaGLMapBufferObject((void**)&cr, vbo);

		dim3 block(8, 8, 1);
		dim3 grid(width / block.x, height / block.y, 1);
		renderKernel << <grid, block >> >(mesh, _start_tet, cam_o, cam_d, cam_u, accumulatebuffer, cr, WangHash(frames), frames);
		gpuErrchk(hipDeviceSynchronize());

		cudaGLUnmapBufferObject(vbo);
		glBindBuffer(GL_ARRAY_BUFFER, vbo);
		glVertexPointer(2, GL_FLOAT, 12, 0);
		glColorPointer(4, GL_UNSIGNED_BYTE, 12, (GLvoid*)8);
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_COLOR_ARRAY);
		glDrawArrays(GL_POINTS, 0, width * height);
		glDisableClientState(GL_VERTEX_ARRAY);

		glfwSwapBuffers(window);
	}
}

int main(int argc, char *argv[])
{
	hipDeviceProp_t  prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	hipChooseDevice(&dev, &prop);

	tetrahedra_mesh tetmesh;
	tetmesh.load_tet_ele("test1.1.ele");
	tetmesh.load_tet_neigh("test1.1.neigh");
	tetmesh.load_tet_node("test1.1.node");
	tetmesh.load_tet_face("test1.1.face");
	tetmesh.load_tet_t2f("test1.1.t2f");


	// ===========================
	//     mesh2
	// ===========================

	gpuErrchk(hipMallocManaged(&mesh, sizeof(mesh2)));

	// INDICES
	mesh->edgenum = tetmesh.edgenum;
	mesh->facenum = tetmesh.facenum;
	mesh->nodenum = tetmesh.nodenum;
	mesh-> tetnum = tetmesh.tetnum;

	// NODES
	hipMallocManaged(&mesh->n_index, mesh->nodenum*sizeof(uint32_t));
	for (auto i : tetmesh.nodes) mesh->n_index[i.index] = i.index;
	hipMallocManaged(&mesh->n_x, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_y, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_z, mesh->nodenum*sizeof(float));
	for (auto i : tetmesh.nodes) mesh->n_x[i.index] = i.x;
	for (auto i : tetmesh.nodes) mesh->n_y[i.index] = i.y;
	for (auto i : tetmesh.nodes) mesh->n_z[i.index] = i.z;

	// FACES
	hipMallocManaged(&mesh->f_index, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_index[i.index] = i.index;
	hipMallocManaged(&mesh->f_node_a, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_b, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_c, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_node_a[i.index] = i.node_a;
	for (auto i : tetmesh.faces) mesh->f_node_b[i.index] = i.node_b;
	for (auto i : tetmesh.faces) mesh->f_node_c[i.index] = i.node_c;
	hipMallocManaged(&mesh->face_is_constrained, mesh->facenum*sizeof(bool));
	hipMallocManaged(&mesh->face_is_wall, mesh->facenum*sizeof(bool));
	for (auto i : tetmesh.faces) mesh->face_is_constrained[i.index] = i.face_is_constrained;
	for (auto i : tetmesh.faces) mesh->face_is_wall[i.index] = i.face_is_wall;

	// TETRAHEDRA
	hipMallocManaged(&mesh->t_index, mesh->tetnum*sizeof(uint32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_index[i.number] = i.number;
	hipMallocManaged(&mesh->t_findex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_findex1[i.number] = i.findex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex2[i.number] = i.findex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex3[i.number] = i.findex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex4[i.number] = i.findex4;
	hipMallocManaged(&mesh->t_nindex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex1[i.number] = i.nindex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex2[i.number] = i.nindex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex3[i.number] = i.nindex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex4[i.number] = i.nindex4;
	hipMallocManaged(&mesh->t_adjtet1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet1[i.number] = i.adjtet1;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet2[i.number] = i.adjtet2;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet3[i.number] = i.adjtet3;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet4[i.number] = i.adjtet4;

	// ===========================
	//     mesh end
	// ===========================

	// Get bounding box
	box = init_BBox(mesh);
	fprintf_s(stderr, "\nBounding box:MIN xyz - %f %f %f \n", box.min.x, box.min.y, box.min.z);
	fprintf_s(stderr, "             MAX xyz - %f %f %f \n\n", box.max.x, box.max.y, box.max.z);

	// Allocate unified memory
	gpuErrchk(hipMallocManaged(&cr, width * height * sizeof(float3)));
	gpuErrchk(hipMallocManaged(&accumulatebuffer, width * height * sizeof(float3)));

	// grid dimensions for finding starting tetrahedra
	uint32_t _dim = 2+pow(mesh->tetnum, 0.25);
	dim3 Block(_dim, _dim, 1);
	dim3 Grid(_dim, _dim, 1);
	GetTetrahedraFromPoint << <Grid, Block >> >(mesh, cam_o);
	gpuErrchk(hipDeviceSynchronize()); 

	if (_start_tet == 0) 
	{
		fprintf(stderr, "Starting point outside tetrahedra! Aborting ... \n");
		system("PAUSE");
		exit(0);

	} else fprintf(stderr, "Starting tetrahedra - camera: %lu \n", _start_tet);
	
	// main render function

	render();

	gpuErrchk(hipDeviceReset());
	glfwTerminate();
	return 0;
}


