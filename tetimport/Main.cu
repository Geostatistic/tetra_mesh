#include "hip/hip_runtime.h"
/*
*  tetrahedra-based raytracer
*  Copyright (C) 2015  Christian Lehmann
*
*  This program is free software; you can redistribute it and/or modify
*  it under the terms of the GNU General Public License as published by
*  the Free Software Foundation; either version 2 of the License, or
*  (at your option) any later version.
*
*  This program is distributed in the hope that it will be useful,
*  but WITHOUT ANY WARRANTY; without even the implied warranty of
*  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*  GNU General Public License for more details.
*/

#define GLEW_STATIC
#include <stdio.h>
#include "tetgen_io.h"
#include "cuPrintf.cuh"
#include ""
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v7.5\extras\CUPTI\include\GL\glew.h"
#include "GLFW/glfw3.h"
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int width = 640, height=480, spp = 4;
float3* cr;
int frames = 0;
__device__ float gamma = 2.2f;
__device__ float fov = 45.0f;
BBox box;
GLuint vbo;
mesh2 *mesh;

// Camera
bool    keys[1024];
GLfloat sensitivity = 0.1f;
bool firstMouse = true;
float4 cam_o = make_float4(7, -8, -4, 0);
float4 cam_d = make_float4(0, 0, 0, 0);
float4 cam_u = make_float4(0, 0, 1, 0);
GLfloat Yaw = 90.0f;	// horizontal inclination
GLfloat Pitch = 0.0f; // vertikal inclination
GLfloat lastX = width / 2.0; //screen center
GLfloat lastY = height / 2.0;
GLfloat deltaTime = 0.0f;	// Time between current frame and last frame
GLfloat lastFrame = 0.0f;

union Color  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};


// CUDA error checking
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		system("PAUSE");
		if (abort) exit(code);
	}
}

unsigned int WangHash(unsigned int a) {
	// richiesams.blogspot.co.nz/2015/03/creating-randomness-and-acummulating.html
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

static void error_callback(int error, const char* description)
{
	fputs(description, stderr);
}


static void key_callback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
	// first value sets velocity
	GLfloat cameraSpeed = 3.5f * deltaTime;
	if (key >= 0 && key < 1024)
	{
		if (action == GLFW_PRESS)
		{
			keys[key] = true;
			//look for new tetrahedra...
			uint32_t _dim = 2 + pow(mesh->tetnum, 0.25);
			dim3 Block(_dim, _dim, 1);
			dim3 Grid(_dim, _dim, 1);
			GetTetrahedraFromPoint << <Grid, Block >> >(mesh, cam_o);
			gpuErrchk(hipDeviceSynchronize());
		}
		else if (action == GLFW_RELEASE)
		{
			keys[key] = false;
			uint32_t _dim = 2 + pow(mesh->tetnum, 0.25);
			dim3 Block(_dim, _dim, 1);
			dim3 Grid(_dim, _dim, 1);
			GetTetrahedraFromPoint << <Grid, Block >> >(mesh, cam_o);
			gpuErrchk(hipDeviceSynchronize());
		}

	}
	if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
	{
		glfwSetWindowShouldClose(window, GL_TRUE);
	}
		
	if (keys[GLFW_KEY_A])
	{
		cam_o -= normalizeCPU(CrossCPU(cam_d, cam_u)) % cameraSpeed;
	}
	if (keys[GLFW_KEY_D])
	{
		cam_o += normalizeCPU(CrossCPU(cam_d, cam_u)) % cameraSpeed;
	}
	if (keys[GLFW_KEY_W])
	{
		cam_o += cam_d % cameraSpeed;
	}
	if (keys[GLFW_KEY_S])
	{
		cam_o -= cam_d % cameraSpeed;
	}
}

void mouse_callback(GLFWwindow* window, double xpos, double ypos)
{	
	if (firstMouse)
	{
		lastX = xpos;
		lastY = ypos;
		firstMouse = false;
	}
	GLfloat xoffset = xpos - lastX;
	GLfloat yoffset = lastY - ypos; // Reversed since y-coordinates go from bottom to left
	
	lastX = xpos;
	lastY = ypos;
	xoffset *= sensitivity;
	yoffset *= sensitivity;

	Yaw += xoffset;
	Pitch += yoffset;

	// Make sure that when pitch is out of bounds, screen doesn't get flipped
	if (Pitch > 89.0f)
		Pitch = 89.0f;
	if (Pitch < -89.0f)
		Pitch = -89.0f;

	float4 front;
	float4 cam_r;
	front.x = cos(radian(Yaw)) * cos(radian(Pitch));
	front.y = sin(radian(Pitch));
	front.z = sin(radian(Yaw)) * cos(radian(Pitch));
	cam_d = normalizeCPU(front);
	//cam_r = normalizeCPU(CrossCPU(cam_d, cam_u));
	//cam_u = normalizeCPU(CrossCPU(cam_r, cam_d));
}



__device__ float getDepth(Ray r, mesh2 *mesh, rayhit firsthit)
{
	float4 a1 = make_float4(mesh->n_x[mesh->f_node_a[firsthit.face]], mesh->n_y[mesh->f_node_a[firsthit.face]], mesh->n_z[mesh->f_node_a[firsthit.face]], 0);
	float4 a2 = make_float4(mesh->n_x[mesh->f_node_b[firsthit.face]], mesh->n_y[mesh->f_node_b[firsthit.face]], mesh->n_z[mesh->f_node_b[firsthit.face]], 0);
	float4 a3 = make_float4(mesh->n_x[mesh->f_node_c[firsthit.face]], mesh->n_y[mesh->f_node_c[firsthit.face]], mesh->n_z[mesh->f_node_c[firsthit.face]], 0);
	float c = abs(intersect_dist(r, a1, a2, a3));
	float new_value = ((c - 0.f) / (60.f - 0.f)) * (1.f - 0.f) + 0.f;
	return new_value;
}


__device__ RGB radiance(Ray r, mesh2 *mesh, int32_t start, int depth)
{
	rayhit firsthit;
	traverse_ray(mesh, r, start, firsthit, depth);
	float d2 = getDepth(r, mesh, firsthit); // gets depth value
	RGB rd;
	rd.x = 0; rd.y = 0; rd.z = d2;
	return rd; 
}


__global__ void renderKernel(mesh2 *tetmesh, int32_t start, float4 cam_o, float4 cam_d, float4 cam_u, float3 *c, unsigned int hashedframenumber)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = (height - y - 1)*width + x;

	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprandState randState;
	hiprand_init(hashedframenumber + threadId, 0, 0, &randState);

	RGB c0(0,0,0);
	for (int s = 0; s < spp; s++)
	{
		float yu = 1.0f - ((y + hiprand_uniform(&randState)) / float(height - 1));
		float xu = (x + hiprand_uniform(&randState)) / float(width - 1);
		Ray _ray = makeCameraRay(fov, cam_o, cam_d, cam_u, xu, yu);
		RGB rd = radiance(_ray, tetmesh, start, 0);
		c0 = c0 + rd;
	}
	c0 = c0 / 4;

	Color fcolour;
	float3 colour = make_float3(clamp(c0.x, 0.0f, 1.0f), clamp(c0.y, 0.0f, 1.0f), clamp(c0.z, 0.0f, 1.0f));

	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / gamma) * 255), (unsigned char)(powf(colour.y, 1 / gamma) * 255), (unsigned char)(powf(colour.z, 1 / gamma) * 255), 1);
	c[i] = make_float3(x, y, fcolour.c);
}


void render()
{
	GLFWwindow* window;
	if (!glfwInit()) exit(EXIT_FAILURE);
	window = glfwCreateWindow(width, height, "tetra_mesh", NULL, NULL);
	glfwMakeContextCurrent(window);
	glfwSetErrorCallback(error_callback);
	glfwSetKeyCallback(window, key_callback);
	glfwSetCursorPosCallback(window, mouse_callback);
	glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);

	glewExperimental = GL_TRUE;
	glewInit();
	if (!glewIsSupported("GL_VERSION_2_0 ")) 
	{
		fprintf(stderr, "GLEW not supported.");
		fflush(stderr);
		exit(0);
	}
	fprintf(stderr, "GLEW successfully initialized  \n");


	glClearColor(0.0, 0.0, 0.0, 0.0);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glOrtho(0.0, width, 0.0, height, 0, 1);
	
	glGenBuffers(1, &vbo);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glBufferData(GL_ARRAY_BUFFER, width * height * sizeof(float3), 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	cudaGLRegisterBufferObject(vbo);
	fprintf(stderr, "VBO created  \n");
	fprintf(stderr, "Entering glutMainLoop...  \n");

	while (!glfwWindowShouldClose(window))
	{
		// Calculate deltatime of current frame
		GLfloat currentFrame = glfwGetTime();
		deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;
		glfwPollEvents();

		//frames++;
		cudaGLMapBufferObject((void**)&cr, vbo);
		glClear(GL_COLOR_BUFFER_BIT);

		dim3 block(8, 8, 1);
		dim3 grid(width / block.x, height / block.y, 1);
		renderKernel << <grid, block >> >(mesh, _start_tet, cam_o, cam_d, cam_u, cr, WangHash(frames));
		gpuErrchk(hipDeviceSynchronize());

		cudaGLUnmapBufferObject(vbo);
		glBindBuffer(GL_ARRAY_BUFFER, vbo);
		glVertexPointer(2, GL_FLOAT, 12, 0);
		glColorPointer(4, GL_UNSIGNED_BYTE, 12, (GLvoid*)8);
		glEnableClientState(GL_VERTEX_ARRAY);
		glEnableClientState(GL_COLOR_ARRAY);
		glDrawArrays(GL_POINTS, 0, width * height);
		glDisableClientState(GL_VERTEX_ARRAY);

		glfwSwapBuffers(window);
	}
}

int main(int argc, char *argv[])
{
	hipDeviceProp_t  prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	hipChooseDevice(&dev, &prop);

	tetrahedra_mesh tetmesh;
	tetmesh.load_tet_ele("test2.1.ele");
	tetmesh.load_tet_neigh("test2.1.neigh");
	tetmesh.load_tet_node("test2.1.node");
	tetmesh.load_tet_face("test2.1.face");
	tetmesh.load_tet_t2f("test2.1.t2f");


	// ===========================
	//     mesh2
	// ===========================

	gpuErrchk(hipMallocManaged(&mesh, sizeof(mesh2)));

	// INDICES
	mesh->edgenum = tetmesh.edgenum;
	mesh->facenum = tetmesh.facenum;
	mesh->nodenum = tetmesh.nodenum;
	mesh-> tetnum = tetmesh.tetnum;

	// NODES
	hipMallocManaged(&mesh->n_index, mesh->nodenum*sizeof(uint32_t));
	for (auto i : tetmesh.nodes) mesh->n_index[i.index] = i.index;
	hipMallocManaged(&mesh->n_x, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_y, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_z, mesh->nodenum*sizeof(float));
	for (auto i : tetmesh.nodes) mesh->n_x[i.index] = i.x;
	for (auto i : tetmesh.nodes) mesh->n_y[i.index] = i.y;
	for (auto i : tetmesh.nodes) mesh->n_z[i.index] = i.z;

	// FACES
	hipMallocManaged(&mesh->f_index, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_index[i.index] = i.index;
	hipMallocManaged(&mesh->f_node_a, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_b, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_c, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_node_a[i.index] = i.node_a;
	for (auto i : tetmesh.faces) mesh->f_node_b[i.index] = i.node_b;
	for (auto i : tetmesh.faces) mesh->f_node_c[i.index] = i.node_c;
	hipMallocManaged(&mesh->face_is_constrained, mesh->facenum*sizeof(bool));
	hipMallocManaged(&mesh->face_is_wall, mesh->facenum*sizeof(bool));
	for (auto i : tetmesh.faces) mesh->face_is_constrained[i.index] = i.face_is_constrained;
	for (auto i : tetmesh.faces) mesh->face_is_wall[i.index] = i.face_is_wall;

	// TETRAHEDRA
	hipMallocManaged(&mesh->t_index, mesh->tetnum*sizeof(uint32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_index[i.number] = i.number;
	hipMallocManaged(&mesh->t_findex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_findex1[i.number] = i.findex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex2[i.number] = i.findex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex3[i.number] = i.findex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex4[i.number] = i.findex4;
	hipMallocManaged(&mesh->t_nindex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex1[i.number] = i.nindex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex2[i.number] = i.nindex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex3[i.number] = i.nindex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex4[i.number] = i.nindex4;
	hipMallocManaged(&mesh->t_adjtet1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet1[i.number] = i.adjtet1;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet2[i.number] = i.adjtet2;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet3[i.number] = i.adjtet3;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet4[i.number] = i.adjtet4;

	// ===========================
	//     mesh end
	// ===========================

	// Get bounding box
	box = init_BBox(mesh);
	fprintf_s(stderr, "\nBounding box:MIN xyz - %f %f %f \n", box.min.x, box.min.y, box.min.z);
	fprintf_s(stderr, "             MAX xyz - %f %f %f \n\n", box.max.x, box.max.y, box.max.z);

	// Get camera starting tetrahedra
	gpuErrchk(hipMallocManaged(&cr, width * height * sizeof(float3)));

	// grid dimensions for finding starting tetrahedra
	uint32_t _dim = 2+pow(mesh->tetnum, 0.25);
	dim3 Block(_dim, _dim, 1);
	dim3 Grid(_dim, _dim, 1);
	GetTetrahedraFromPoint << <Grid, Block >> >(mesh, cam_o);
	gpuErrchk(hipDeviceSynchronize()); 

	if (_start_tet == 0) 
	{
		fprintf(stderr, "Starting point outside tetrahedra! Aborting ... \n");
		system("PAUSE");
		exit(0);

	} else fprintf(stderr, "Starting tetrahedra - camera: %lu \n", _start_tet);
	
	// main render function

	render();

	gpuErrchk(hipDeviceReset());
	glfwTerminate();
	return 0;
}


