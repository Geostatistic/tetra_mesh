#include "hip/hip_runtime.h"
#define GLEW_STATIC
#include "tetgen_io.h"
#include "cuPrintf.cuh"
#include ""
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v7.5\extras\CUPTI\include\GL\glew.h"
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v7.5\extras\CUPTI\include\GL\glut.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

const int width = 320, height=240, spp = 4;
float3*cr;
float3* accumulatebuffer;
int frames = 0;
GLuint vbo;
void *d_vbo_buffer = NULL;
mesh2 *mesh;

float4 cam_o = make_float4(-16, 5, -5, 0);
float4 cam_d = make_float4(0, 0, 0, 0);
float4 cam_u = make_float4(0, 0, 1, 0);

union Color  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};

void Timer(int obsolete) {

	glutPostRedisplay();
	glutTimerFunc(30, Timer, 0);
}

__device__ float timer = 0.0f;



unsigned int WangHash(unsigned int a) {
	a = (a ^ 61) ^ (a >> 16);
	a = a + (a << 3);
	a = a ^ (a >> 4);
	a = a * 0x27d4eb2d;
	a = a ^ (a >> 15);
	return a;
}

// CUDA error checking
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		system("PAUSE");
		if (abort) exit(code);
	}
}


__device__ float getDepth(Ray r, mesh2 *mesh, rayhit firsthit)
{
	float4 a1 = make_float4(mesh->n_x[mesh->f_node_a[firsthit.face]], mesh->n_y[mesh->f_node_a[firsthit.face]], mesh->n_z[mesh->f_node_a[firsthit.face]], 0);
	float4 a2 = make_float4(mesh->n_x[mesh->f_node_b[firsthit.face]], mesh->n_y[mesh->f_node_b[firsthit.face]], mesh->n_z[mesh->f_node_b[firsthit.face]], 0);
	float4 a3 = make_float4(mesh->n_x[mesh->f_node_c[firsthit.face]], mesh->n_y[mesh->f_node_c[firsthit.face]], mesh->n_z[mesh->f_node_c[firsthit.face]], 0);
	float c = abs(intersect_dist(r, a1, a2, a3));
	//float k = ((255 - 0) / (0 - 80)); // in zweiter klammer erster wert ist untere grenze distanzwerte
	//float d = 0 - (80 * k);
	//return (c*k) + d;
	float new_value = ((c - 0.f) / (80.f - 0.f)) * (1.f - 0.f) + 0.f;

	return new_value;
}


__device__ RGB radiance(Ray r, mesh2 *mesh, int32_t start, int depth)
{
	rayhit firsthit;
	traverse_ray(mesh, r, start, firsthit, depth);
	float d2 = getDepth(r, mesh, firsthit); // return depth value
	RGB rd;
	rd.x = 0; rd.y = 0; rd.z = d2;
	return rd; 
}


__global__ void renderKernel(mesh2 *tetmesh, int32_t start, float4 cam_o, float4 cam_d, float4 cam_u, float3 *c, unsigned int hashedframenumber)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int i = (height - y - 1)*width + x;

	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedframenumber + threadId, 0, 0, &randState);

	// raytracing stuff
	RGB c0(0);

	for (int s = 0; s < spp; s++)
			{
		float yu = 1.0f - ((y + hiprand_uniform(&randState)) / float(height - 1));
		float xu = (x + hiprand_uniform(&randState)) / float(width - 1);
				Ray ray = makeCameraRay(45.0f, cam_o, cam_d, cam_u, xu, yu);
				RGB rd = radiance(ray, tetmesh, start, 0);
				c0 = c0 + rd;
			}
	c0 = c0 / 4;

	Color fcolour;
	float3 colour = make_float3(clamp(c0.x, 0.0f, 1.0f), clamp(c0.y, 0.0f, 1.0f), clamp(c0.z, 0.0f, 1.0f));
	// convert from 96-bit to 24-bit colour + perform gamma correction
	fcolour.components = make_uchar4((unsigned char)(powf(colour.x, 1 / 2.2f) * 255), (unsigned char)(powf(colour.y, 1 / 2.2f) * 255), (unsigned char)(powf(colour.z, 1 / 2.2f) * 255), 1);
	
	//fcolour.components = make_uchar4((unsigned char)(powf(0.1f, 1 / 2.2f) * 255), (unsigned char)(powf(0.7f, 1 / 2.2f) * 255), (unsigned char)(powf(0.4f, 1 / 2.2f) * 255), 1);

	
	// store pixel coordinates and pixelcolour in OpenGL readable outputbuffer
	c[i] = make_float3(x, y, fcolour.c);




	//c[i] = make_float4( c0.x, c0.y, c0.z,0);
}


void disp(void)
{
	frames++;
	hipDeviceSynchronize();
	cudaGLMapBufferObject((void**)&cr, vbo);
	glClear(GL_COLOR_BUFFER_BIT);

	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);
	//20
	renderKernel << <grid, block >> >(mesh, start, cam_o, cam_d, cam_u, cr, WangHash(frames));
	//renderKernel << <1,1 >> >(mesh, start, cam_o, cam_d, cam_u, cr);
	gpuErrchk(hipDeviceSynchronize());

	cudaGLUnmapBufferObject(vbo);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(2, GL_FLOAT, 12, 0);
	glColorPointer(4, GL_UNSIGNED_BYTE, 12, (GLvoid*)8);

	glEnableClientState(GL_VERTEX_ARRAY);
	glEnableClientState(GL_COLOR_ARRAY);
	glDrawArrays(GL_POINTS, 0, width * height);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();
	//glutPostRedisplay();
}

void render()
{

	hipMalloc(&accumulatebuffer, width * height * sizeof(float3));
	char *argv[] = { "null", NULL };
	int   argc = 1;
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
	glutInitWindowPosition(100, 100);
	glutInitWindowSize(width, height);
	glutCreateWindow("tetra_mesh");
	glClearColor(0.0, 0.0, 0.0, 0.0);
	glMatrixMode(GL_PROJECTION);
	gluOrtho2D(0.0, width, 0.0, height);
	fprintf(stderr, "OpenGL successfully initialized \n");
	glutDisplayFunc(disp);
	glewInit();
	if (!glewIsSupported("GL_VERSION_2_0 ")) {
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		exit(0);
	}
	fprintf(stderr, "GLEW successfully initialized  \n");
	Timer(0);
	glGenBuffers(1, &vbo);
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	unsigned int size = width * height * sizeof(float3);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	cudaGLRegisterBufferObject(vbo);
	fprintf(stderr, "VBO created  \n");
	fprintf(stderr, "Entering glutMainLoop...  \n");
	glutMainLoop();
}








int main(int argc, char *argv[])
{
	int frames = 0;
	hipDeviceProp_t  prop;
	int dev;
	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	hipChooseDevice(&dev, &prop);

	tetrahedra_mesh tetmesh;
	tetmesh.load_tet_ele("test1.1.ele");
	tetmesh.load_tet_neigh("test1.1.neigh");
	tetmesh.load_tet_node("test1.1.node");
	tetmesh.load_tet_face("test1.1.face");
	tetmesh.load_tet_t2f("test1.1.t2f");



	
	// ===========================
	//     mesh2 testing
	// ===========================


	gpuErrchk(hipMallocManaged(&mesh, sizeof(mesh2)));

	// ELEMENT INDICES
	mesh->edgenum = tetmesh.edgenum;
	mesh->facenum = tetmesh.facenum;
	mesh->nodenum = tetmesh.nodenum;
	mesh-> tetnum = tetmesh.tetnum;

	// NODES - funktioniert
	hipMallocManaged(&mesh->n_index, mesh->nodenum*sizeof(uint32_t));
	for (auto i : tetmesh.nodes) mesh->n_index[i.index] = i.index;
	hipMallocManaged(&mesh->n_x, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_y, mesh->nodenum*sizeof(float));
	hipMallocManaged(&mesh->n_z, mesh->nodenum*sizeof(float));
	for (auto i : tetmesh.nodes) mesh->n_x[i.index] = i.x;
	for (auto i : tetmesh.nodes) mesh->n_y[i.index] = i.y;
	for (auto i : tetmesh.nodes) mesh->n_z[i.index] = i.z;

	// FACES
	hipMallocManaged(&mesh->f_index, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_index[i.index] = i.index;
	hipMallocManaged(&mesh->f_node_a, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_b, mesh->facenum*sizeof(uint32_t));
	hipMallocManaged(&mesh->f_node_c, mesh->facenum*sizeof(uint32_t));
	for (auto i : tetmesh.faces) mesh->f_node_a[i.index] = i.node_a;
	for (auto i : tetmesh.faces) mesh->f_node_b[i.index] = i.node_b;
	for (auto i : tetmesh.faces) mesh->f_node_c[i.index] = i.node_c;
	hipMallocManaged(&mesh->face_is_constrained, mesh->facenum*sizeof(bool));
	hipMallocManaged(&mesh->face_is_wall, mesh->facenum*sizeof(bool));
	for (auto i : tetmesh.faces) mesh->face_is_constrained[i.index] = i.face_is_constrained;
	for (auto i : tetmesh.faces) mesh->face_is_wall[i.index] = i.face_is_wall;

	// TETRAHEDRA
	hipMallocManaged(&mesh->t_index, mesh->tetnum*sizeof(uint32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_index[i.number] = i.number;
	hipMallocManaged(&mesh->t_findex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_findex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_findex1[i.number] = i.findex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex2[i.number] = i.findex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex3[i.number] = i.findex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_findex4[i.number] = i.findex4;
	hipMallocManaged(&mesh->t_nindex1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_nindex4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex1[i.number] = i.nindex1;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex2[i.number] = i.nindex2;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex3[i.number] = i.nindex3;
	for (auto i : tetmesh.tetrahedras) mesh->t_nindex4[i.number] = i.nindex4;
	hipMallocManaged(&mesh->t_adjtet1, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet2, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet3, mesh->tetnum*sizeof(int32_t));
	hipMallocManaged(&mesh->t_adjtet4, mesh->tetnum*sizeof(int32_t));
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet1[i.number] = i.adjtet1;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet2[i.number] = i.adjtet2;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet3[i.number] = i.adjtet3;
	for (auto i : tetmesh.tetrahedras) mesh->t_adjtet4[i.number] = i.adjtet4;

	// ===========================
	//     mesh end
	// ===========================

	// Get bounding box
	BBox box = init_BBox(mesh);
	fprintf_s(stderr, "\nBounding box:MIN xyz - %f %f %f \n", box.min.x, box.min.y, box.min.z);
	fprintf_s(stderr, "             MAX xyz - %f %f %f \n\n", box.max.x, box.max.y, box.max.z);




	gpuErrchk(hipMallocManaged(&cr, width * height * sizeof(float3)));


	clock_t t1 = clock();


	GetTetrahedraFromPoint << <mesh->tetnum, 1>> >(mesh, cam_o);
	gpuErrchk(hipDeviceSynchronize()); // kamera erfolgreich abgerufen..

	if (start == 0) 
	{
		fprintf(stderr, "Starting point outside tetrahedra! Aborting ... \n");
		system("PAUSE");
		abort;

	} else fprintf(stderr, "Starting tetrahedra - camera: %lu \n", start);
	
	render();


	clock_t t2 = clock();
	double t = (double)(t2 - t1) / CLOCKS_PER_SEC;
	printf("\nRender time: %fs.\n", t);

	FILE *f = fopen("image.ppm", "w");
	fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);
	for (int i = 0; i<width*height; i++)
		fprintf(f, "%d %d %d ", (int)cr[i].x, (int)cr[i].y, (int)cr[i].z);



	

	hipFree(mesh);
	hipFree(cr);

	system("PAUSE");;
}


